#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL(x) {const hipError_t a=(x); if(a!=hipSuccess) {printf("\nCUDA Error: %s (err_num=%d)\n", hipGetErrorString(a), a); exit(1);}}

__global__ void matrixMultiplyKernel(double *d_input, 
                                    double *d_weight, 
                                    double *d_output, 
                                    int inputSize, 
                                    int outputSize) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < outputSize) {
        double sum = 0.0;
        for (int j = 0; j < inputSize; j++) {
            sum += d_input[j] * d_weight[j * outputSize + row];
        }
        d_output[row] = sum;
    }
}

__global__ void sigmoidKernel(double *d_data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_data[idx] = 1.0 / (1.0 + exp(-d_data[idx]));
    }
}

double* layerCreation(int n) {
    double* layer = (double*)malloc(sizeof(double) * n);
    return layer;
}

double** weightTableCreation(int numberOfOutputNode, int numberOfInputNode) {
    double** weightTable = (double**)malloc(sizeof(double*) * numberOfOutputNode);
    for (int i = 0; i < numberOfOutputNode; i++) {
        weightTable[i] = (double*)malloc(sizeof(double) * numberOfInputNode);
        for (int j = 0; j < numberOfInputNode; j++) {
            weightTable[i][j] = (double)rand() / RAND_MAX - 0.5;
        }
    }
    return weightTable;
}

int main() {
    srand(time(NULL));
    int L = 16;
    double** network = (double**)malloc(sizeof(double*) * L);

    int* N = (int*)malloc(sizeof(int) * L);
    for (int i = 0; i < L; i++) {
        N[i] = 256;
        network[i] = layerCreation(N[i]);
    }

    double*** weight = (double***)malloc(sizeof(double**) * (L - 1));
    for (int i = 0; i < (L - 1); i++) {
        weight[i] = weightTableCreation(N[i], N[i + 1]);
    }

    for (int i = 0; i < N[0]; i++) {
        network[0][i] = (double)rand() / RAND_MAX - 0.5;
    }

    // CUDA 관련 변수 선언
    double *d_input, *d_output, **d_weightList;
    CUDA_CALL(hipMalloc((void**)&d_input, sizeof(double) * N[0]));
    CUDA_CALL(hipMalloc((void**)&d_output, sizeof(double) * N[1]));

    // 가중치 배열을 위한 포인터 배열 생성 및 데이터 복사
    d_weightList = (double**)malloc(sizeof(double*) * (L - 1));
    for (int layer = 0; layer < (L - 1); layer++) {
        int inputSize = N[layer];
        int outputSize = N[layer + 1];

        // 가중치 2차원 배열을 1차원 배열로 변환 후 디바이스로 복사
        double *weightFlat = (double*)malloc(sizeof(double) * inputSize * outputSize);
        for (int i = 0; i < inputSize; i++) {
            for (int j = 0; j < outputSize; j++) {
                weightFlat[i * outputSize + j] = weight[layer][j][i];
            }
        }
        CUDA_CALL(hipMalloc((void**)&d_weightList[layer], sizeof(double) * inputSize * outputSize));
        CUDA_CALL(hipMemcpy(d_weightList[layer], weightFlat, sizeof(double) * inputSize * outputSize, hipMemcpyHostToDevice));
        free(weightFlat);
    }

    // 연산 시간 측정
    clock_t start = clock();

    for (int layer = 0; layer < (L - 1); layer++) {
        int inputSize = N[layer];
        int outputSize = N[layer + 1];

        // 입력 레이어를 디바이스로 복사
        CUDA_CALL(hipMemcpy(d_input, network[layer], sizeof(double) * inputSize, hipMemcpyHostToDevice));

        // 행렬 곱셈을 위한 CUDA 커널 호출
        int blockSize = 256;
        int gridSize = (outputSize + blockSize - 1) / blockSize;
        matrixMultiplyKernel<<<gridSize, blockSize>>>(d_input, d_weightList[layer], d_output, inputSize, outputSize);
        CUDA_CALL(hipDeviceSynchronize());

        // 시그모이드 활성화 함수를 위한 CUDA 커널 호출
        sigmoidKernel<<<gridSize, blockSize>>>(d_output, outputSize);
        CUDA_CALL(hipDeviceSynchronize());

        // 결과를 호스트로 복사
        CUDA_CALL(hipMemcpy(network[layer + 1], d_output, sizeof(double) * outputSize, hipMemcpyDeviceToHost));
    }

    clock_t end = clock();
    double duration = (double)(end - start) / CLOCKS_PER_SEC;

    printf("Execution time: %lf seconds\n", duration);

    // CUDA 메모리 해제
    for (int layer = 0; layer < (L - 1); layer++) {
        CUDA_CALL(hipFree(d_weightList[layer]));
    }
    free(d_weightList);
    CUDA_CALL(hipFree(d_input));
    CUDA_CALL(hipFree(d_output));

    // 메모리 해제
    for (int i = 0; i < (L - 1); i++) {
        for (int j = 0; j < N[i]; j++) free(weight[i][j]);
        free(weight[i]);
    }
    free(weight);

    free(N);

    for (int i = 0; i < L; i++) free(network[i]);
    free(network);

    return 0;
}


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

double* layerCreation(int n){
    double* layer = (double*)malloc(sizeof(double) * n);
    return layer;
}

double** weightTableCreation(int numberOfOutputNode, int numberOfInputNode){
    double** weightTable = (double**)malloc(sizeof(double*) * numberOfOutputNode);
    for(int i=0;i<numberOfOutputNode;i++){
        weightTable[i] = (double*)malloc(sizeof(double) * numberOfInputNode);
        // printf("Enter w_%d,1 to w_%d,%d\n",i+1,i+1,numberOfInputNode);
        for(int j=0;j<numberOfInputNode;j++){
            // scanf("%lf",&weightTable[i][j]);
            weightTable[i][j]=(double)rand()/RAND_MAX - 0.5;
        }
    }
    return weightTable;
}

void matrixMultiplication(double* input,
                            int inputSize, 
                            double** weight, 
                            double* output,
                            int outputSize){
    for(int i=0;i<outputSize;i++){
        double add=0.0;
        for(int j=0;j<inputSize;j++){
            add = add + input[j] * weight[j][i];
        }
        output[i] = add;
    }
}

double sigmoid(double x){
    return 1.0 / (1.0 + exp(-x));
}

int main(){
    srand(time(NULL));

    int L;
    // printf("How many layers? Maximum 16\n");
    // scanf("%d",&L);
    L=16;
    double** network = (double**)malloc(sizeof(double*) * L);

    int* N = (int*)malloc(sizeof(int)*L);
    for(int i=0;i<L;i++){
        // printf("How many nodes in layer %d? Maximum 256\n",i+1);
        // scanf("%d",&N[i]);
        N[i]=256;
        network[i] = layerCreation(N[i]);
    }
    
    double*** weight = (double***)malloc(sizeof(double**) * (L-1));
    for(int i=0;i<(L-1);i++){
        // printf("Create weight table for layer %d to %d\n",i+1,i+2);
        weight[i] = weightTableCreation(N[i],N[i+1]);
    }



    // printf("Enter %d input\n",N[0]);
    for(int i=0;i<N[0];i++){
        // scanf("%lf",&network[0][i]);
        network[0][i]=(double)rand()/RAND_MAX - 0.5;
    }

    clock_t start = clock();

    for(int i=0;i<(L-1);i++){
        matrixMultiplication(network[i],N[i],weight[i],network[i+1],N[i+1]);
        for(int j=0;j<N[i+1];j++)network[i+1][j] = sigmoid(network[i+1][j]);
    }

    clock_t end = clock();

    // printf("print output\n");
    // for(int i=0;i<N[L-1];i++)printf("%f ",network[L-1][i]);

    double duration = (double)(end - start) / CLOCKS_PER_SEC;

    printf("Execution time: %lf seconds\n", duration);

    {
        for(int i=0;i<(L-1);i++){
            for(int j=0;j<N[i];j++)free(weight[i][j]);
            free(weight[i]);
        }
        free(weight);

        free(N);

        for(int i=0;i<L;i++)free(network[i]);
        free(network);
        return 0;
    }
}